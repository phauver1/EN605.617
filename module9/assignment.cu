
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hipfft/hipfft.h>
#include <math.h>
#include <time.h>


#define nStocks 25 // Number of stocks to analyze
#define blockSize 256   // length of each stock's time series
#define nDays 365   // length of each stock's time series

__global__ void init(unsigned int seed, hiprandState_t* states,
        const unsigned int size) {
    const unsigned int thread_idx = (blockIdx.x * blockDim.x) + threadIdx.x;
    int stride = blockDim.x * gridDim.x; // Step over the number of threads in each block/grid
	for (int i=thread_idx; i<size; i+=stride) { // Loop until we've reached the end of the array
        hiprand_init(seed, i, 0, &states[i]);
    }
}

__global__ void randoms(hiprandState_t* states, float* numbers,
        const unsigned int size) {
    const unsigned int thread_idx = (blockIdx.x * blockDim.x) + threadIdx.x;
    int stride = blockDim.x * gridDim.x; // Step over the number of threads in each block/grid
	for (int i=thread_idx; i<size; i+=stride) { // Loop until we've reached the end of the array
        // Generate the random number
        float rand_number = hiprand_normal(&states[i]);
        // Cap the number to -5,5 and return
        numbers[i] = fminf(fmaxf(rand_number, -5.0f), 5.0f);
    }
}

int main() {
    hiprandState_t* states; // Allocate space for the cuRAND states
    hipMalloc((void**) &states, nStocks * nDays * sizeof(hiprandState_t));
    init<<<nStocks, blockSize>>>(time(0), states, nStocks*nDays); // Initialize the random states
    float* gpu_nums; // Allocate an array of floats on the GPU
    hipMalloc((void**) &gpu_nums, nStocks * nDays * sizeof(float));
    randoms<<<nStocks, blockSize>>>(states, gpu_nums, nStocks*nDays); // Generate random numbers

    hipfftHandle plan; // Setup cuFFT plan
    hipfftPlan1d(&plan, nDays, HIPFFT_R2C, nStocks); // Batch 1D FFT for each stock
    hipfftComplex* gpu_freq; // Allocate an array for FFT results
    hipMalloc((void**)&gpu_freq, nStocks*(nDays/2+1)*sizeof(hipfftComplex));
    hipfftExecR2C(plan, gpu_nums, gpu_freq); // Execute FFT
    
    // Copy the results back to the CPU
    hipfftComplex* cpu_freq = new hipfftComplex[nStocks*(nDays/2+1)];
    hipMemcpy(cpu_freq, gpu_freq, nStocks*(nDays/2+1)*sizeof(hipfftComplex),
        hipMemcpyDeviceToHost);
    
    // Print spectrum magnitude for each stock
    for (int i = 0; i < nStocks; i++) {
        printf("Stock %d FFT magnitudes:\n", i);
        for (int j = 0; j < nDays/2 + 1; j++) {
            float re = cpu_freq[i*(nDays/2+1) + j].x;
            float im = cpu_freq[i*(nDays/2+1) + j].y;
            float mag = sqrtf(re*re + im*im)/nDays;
            printf("%f ", mag);
        }
        printf("\n\n");
    }
    
    delete[] cpu_freq; // Delete the CPU copy of the FFT results
    hipfftDestroy(plan); // Delete the cuFFT plan
    hipFree(states); // Delete the random states
    hipFree(gpu_nums); // Delete the cuRAND generated random numbers
    hipFree(gpu_freq); // Delete the GPU copy of the FFT results
    return 0; // End the mainloop
}
